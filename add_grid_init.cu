
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>

// CUDA Kernel function to init elements of two arrays on the GPU
__global__
void init(int n, float *x, float *y) 
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int stride = blockDim.x * gridDim.x;
	for (int i = index; i < n; i += stride) {
		x[i] = 1.0f;
		y[i] = 2.0f;
	}
}

// CUDA Kernel function to add the elements of two arrays on the GPU
__global__
void add(int n, float *x, float *y)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int stride = blockDim.x * gridDim.x;
	for (int i = index; i < n; i += stride) {
		y[i] = x[i] + y[i];
	}
}

int main(void)
{
	int N = 1<<20;

	float *x, *y;
	hipMallocManaged(&x, N*sizeof(float));
	hipMallocManaged(&y, N*sizeof(float));

	// Init x and y arrays and run add kernel on the CPU
	int blockSize = 256;
	int numBlocks = (N + blockSize - 1) / blockSize;
	init<<<numBlocks, blockSize>>>(N, x, y);
	add<<<numBlocks, blockSize>>>(N, x, y);

	hipDeviceSynchronize();

	// Check for errors (all values should be 3.0f)
	float maxError = 0.0f;
	for (int i = 0; i < N; i++) {
		maxError = fmax(maxError, fabs(y[i] - 3.0f));
	}
	std::cout << "Max error: " << maxError << std::endl;

	// Free memory
	hipFree(x);
	hipFree(y);

	return 0;
}
